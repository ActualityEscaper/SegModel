
#include <vector>

#include "caffe/layers/operator/instance_cudnn_batch_norm_layer.hpp"

namespace caffe {


template <typename Dtype>
void InstanceCuDNNBatchNormLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
	for (int n=0;n < bottom[0]->num();n++)
	{
		CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(Caffe::cudnn_handle(gpu_id_),
			    HIPDNN_BATCHNORM_SPATIAL,
			    cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
			    bottom_desc_, bottom[0]->gpu_data() + bottom[0]->offset(n),
			    top_desc_, top[0]->mutable_gpu_data() + top[0]->offset(n),
			    scale_bias_desc_,this->blobs_[0]->gpu_data(),this->blobs_[1]->gpu_data(),
			    Dtype(1),
			    this->blobs_[2]->mutable_gpu_data(),this->blobs_[3]->mutable_gpu_data(),
			    double(HIPDNN_BN_MIN_EPSILON),
			    savedmean.mutable_gpu_data()+savedmean.offset(n),savedinvvariance.mutable_gpu_data()+savedinvvariance.offset(n)));	    
	}   
}

template <typename Dtype>
void InstanceCuDNNBatchNormLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, const vector<Blob<Dtype>*>& bottom) 
{
	
	const Dtype* top_data = top[0]->gpu_data();
	const Dtype* top_diff = top[0]->gpu_diff();
	const Dtype* bottom_data = bottom[0]->gpu_data();
	Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
	for (int n=0;n < bottom[0]->num();n++)
	{
		CUDNN_CHECK(hipdnnBatchNormalizationBackward(Caffe::cudnn_handle(gpu_id_),
					HIPDNN_BATCHNORM_SPATIAL,
					cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
					cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::one,
					bottom_desc_, bottom[0]->gpu_data() + bottom[0]->offset(n),
					top_desc_,top[0]->gpu_diff() + top[0]->offset(n),
					bottom_desc_, bottom[0]->mutable_gpu_diff() + bottom[0]->offset(n),
					scale_bias_desc_,this->blobs_[0]->gpu_data(),this->blobs_[0]->mutable_gpu_diff(),this->blobs_[1]->mutable_gpu_diff(),
					double(HIPDNN_BN_MIN_EPSILON),
					savedmean.gpu_data()+savedmean.offset(n),savedinvvariance.gpu_data()+savedinvvariance.offset(n)));		
	}
}
template <typename Dtype>
void InstanceCuDNNBatchNormLayer<Dtype>::SecForward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
}
INSTANTIATE_LAYER_GPU_FUNCS(InstanceCuDNNBatchNormLayer);

}  // namespace caffe
