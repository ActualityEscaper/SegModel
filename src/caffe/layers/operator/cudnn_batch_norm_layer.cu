#include "hip/hip_runtime.h"
#include <vector>
#include "caffe/layers/operator/cudnn_batch_norm_layer.hpp"
#define BN_EPS Dtype(1e-5)


namespace caffe {
template <typename Dtype>
static __global__ void linear_batch_norm_forward(int num,int channels,int height,int width,
													const Dtype *weight,const Dtype * in, const Dtype * bias, Dtype *out)
{
  CUDA_KERNEL_LOOP(ind,num*channels*height*width)
  {
  	int c = ind / width / height % channels;
  	out[ind] = weight[c] * in[ind] + bias[c];
  }
}

template <typename Dtype>
static __global__ void linear_batch_norm_backward(int num,int channels,int height,int width,
													const Dtype *weight,const Dtype * in, const Dtype * bias, Dtype *out)
{
  CUDA_KERNEL_LOOP(ind,num*channels*height*width)
  {
  	int c = ind / width / height % channels;
  	out[ind] = weight[c] * in[ind];
  }
}

template <typename Dtype>
void CuDNNBatchNormLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
	if (Caffe::bn_state() == "frozen")
	{
		const int K = bottom[0]->channels();
		weights.Reshape(1,K,1,1);
		bias.Reshape(1,K,1,1);
	
		for(int c=0;c<K;c++)
		{
			weights.mutable_cpu_data()[c] = this->blobs_[0]->cpu_data()[c] / (sqrtf(this->blobs_[3]->cpu_data()[c]+ Dtype(HIPDNN_BN_MIN_EPSILON)));
			bias.mutable_cpu_data()[c] = -this->blobs_[0]->cpu_data()[c]*this->blobs_[2]->cpu_data()[c] / (sqrtf(this->blobs_[3]->cpu_data()[c] + Dtype(HIPDNN_BN_MIN_EPSILON)))
																								+this->blobs_[1]->cpu_data()[c];															
		}				
	} 	

	if (Caffe::number_collect_sample == 0 && Caffe::bn_state() == "learned")
	{
		caffe_gpu_set(this->blobs_[2]->count(),Dtype(0),this->blobs_[2]->mutable_gpu_data());
		caffe_gpu_set(this->blobs_[3]->count(),Dtype(0),this->blobs_[3]->mutable_gpu_data());
	}

  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  
   
	if (Caffe::bn_state() == "learned")
	{	
		double factor;
		if (Caffe::number_collect_sample == -1)
			factor = 0.01;
		else 
			factor = double(1)/double(Caffe::number_collect_sample+1);
	

		CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(Caffe::cudnn_handle(gpu_id_),
		      HIPDNN_BATCHNORM_SPATIAL,
		      cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
		      bottom_desc_, bottom_data,
		      top_desc_,top_data,
		      scale_bias_desc_,this->blobs_[0]->gpu_data(),this->blobs_[1]->gpu_data(),
		      factor,
		      this->blobs_[2]->mutable_gpu_data(),this->blobs_[3]->mutable_gpu_data(),
		      double(HIPDNN_BN_MIN_EPSILON),
		      mean_buffer_->mutable_gpu_data(),var_buffer_->mutable_gpu_data()));	     
  }  
	else
	{

		linear_batch_norm_forward<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>
		(bottom[0]->num(),bottom[0]->channels(),bottom[0]->height(),bottom[0]->width(),
		weights.gpu_data(),bottom[0]->gpu_data(),bias.gpu_data(),top[0]->mutable_gpu_data()); 
/*
		CUDNN_CHECK(hipdnnBatchNormalizationForwardInference(Caffe::parallel_cudnn_handle(gpu_id_),
		      HIPDNN_BATCHNORM_SPATIAL,
		      cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
		      bottom_desc_, bottom_data,
		      top_desc_,top_data,
		      scale_bias_desc_,this->blobs_[0]->gpu_data(),this->blobs_[1]->gpu_data(),
		      this->blobs_[2]->mutable_gpu_data(),this->blobs_[3]->mutable_gpu_data(),
		      double(0.001)
		      ));	       	       
*/	         
	}	   	           
}

template <typename Dtype>
void CuDNNBatchNormLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, const vector<Blob<Dtype>*>& bottom) 
{
	if (Caffe::bn_state() == "learned")
  {
		const Dtype* top_data = top[0]->gpu_data();
		const Dtype* top_diff = top[0]->gpu_diff();
		const Dtype* bottom_data = bottom[0]->gpu_data();
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
		if (Caffe::frozen_param() == false)
		{		
			CUDNN_CHECK(hipdnnBatchNormalizationBackward(Caffe::cudnn_handle(gpu_id_),
						HIPDNN_BATCHNORM_SPATIAL,
						cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
						cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::one,
						bottom_desc_, bottom_data,
						top_desc_,top_diff,
						bottom_desc_, bottom_diff,
						scale_bias_desc_,this->blobs_[0]->gpu_data(),this->blobs_[0]->mutable_gpu_diff(),this->blobs_[1]->mutable_gpu_diff(),
						double(HIPDNN_BN_MIN_EPSILON),
						mean_buffer_->mutable_gpu_data(),var_buffer_->mutable_gpu_data()));	   			
		}
		else
		{		
			CUDNN_CHECK(hipdnnBatchNormalizationBackward(Caffe::cudnn_handle(gpu_id_),
						HIPDNN_BATCHNORM_SPATIAL,
						cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
						cudnn::dataType<Dtype>::zero,cudnn::dataType<Dtype>::one,
						bottom_desc_, bottom_data,
						top_desc_,top_diff,
						bottom_desc_, bottom_diff,
						scale_bias_desc_,this->blobs_[0]->gpu_data(),this->blobs_[0]->mutable_gpu_diff(),this->blobs_[1]->mutable_gpu_diff(),//not use
						double(HIPDNN_BN_MIN_EPSILON),
						mean_buffer_->mutable_gpu_data(),var_buffer_->mutable_gpu_data()));	   	
		}
  }    
  else
  {
  	linear_batch_norm_backward<Dtype><<<CAFFE_GET_BLOCKS(top[0]->count()), CAFFE_CUDA_NUM_THREADS>>>
		(bottom[0]->num(),bottom[0]->channels(),bottom[0]->height(),bottom[0]->width(),
		weights.gpu_data(),top[0]->gpu_diff(),bias.gpu_data(),bottom[0]->mutable_gpu_diff());  
  } 
}
template <typename Dtype>
void CuDNNBatchNormLayer<Dtype>::SecForward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
	int num = bottom[0]->num();
  int channels = bottom[0]->channels();
  int height = bottom[0]->height();
  int width = bottom[0]->width();
	
	CUDNN_CHECK(hipdnnBatchNormalizationBackward(Caffe::cudnn_handle(gpu_id_),
						HIPDNN_BATCHNORM_SPATIAL,
						cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
						cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::one,
						bottom_desc_, bottom[0]->gpu_data(),
						bottom_desc_,bottom[0]->gpu_sec_diff(),
						top_desc_, top[0]->mutable_gpu_sec_diff(),
						scale_bias_desc_,this->blobs_[0]->gpu_data(),this->blobs_[0]->mutable_gpu_diff(),this->blobs_[1]->mutable_gpu_sec_diff(),//blobs_[1]->diff shoud be fixed
						double(HIPDNN_BN_MIN_EPSILON),
						mean_buffer_->mutable_gpu_data(),var_buffer_->mutable_gpu_data()));	   
}
INSTANTIATE_LAYER_GPU_FUNCS(CuDNNBatchNormLayer);

}  // namespace caffe
