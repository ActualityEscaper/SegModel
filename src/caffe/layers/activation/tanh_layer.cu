#include "hip/hip_runtime.h"

#include <vector>

#include "caffe/layers/activation/tanh_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
static __global__ void TanHForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = tanh(in[index]);
  }
}

template <typename Dtype>
static __global__ void TanHBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype tanhx = out_data[index];
    out_diff[index] = in_diff[index] * (1 - tanhx * tanhx);
  }
}
template <typename Dtype>
void TanHLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
	const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  TanHForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
  (count, bottom_data, top_data);
}

template <typename Dtype>
void TanHLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, const vector<Blob<Dtype>*>& bottom) 
{

  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  TanHBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
  (count, top_diff, top_data, bottom_diff);
}
template <typename Dtype>
void TanHLayer<Dtype>::SecForward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
}
INSTANTIATE_LAYER_GPU_FUNCS(TanHLayer);
}  // namespace caffe
