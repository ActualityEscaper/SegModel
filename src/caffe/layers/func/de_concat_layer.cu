#include "hip/hip_runtime.h"

#include <vector>

#include "caffe/layers/func/de_concat_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {
template <typename Dtype>
static __global__ void deconcat_forward(int count,int channels, int i_channels, int cur_channels,int spatial_dim,
																const Dtype * b, Dtype *a)
{
	CUDA_KERNEL_LOOP(i, count)
	{
		int n = i / spatial_dim / i_channels;
		int c = i / spatial_dim % i_channels;
		int s = i % spatial_dim;
		
		a[i] = b[(n*channels+cur_channels+c)*spatial_dim+s];		
	}
}

template <typename Dtype>
static __global__ void deconcat_backward(int count,int channels, int i_channels, int cur_channels,int spatial_dim,
																const Dtype *a, Dtype *b)
{
	CUDA_KERNEL_LOOP(i, count)
	{
		int n = i / spatial_dim / i_channels;
		int c = i / spatial_dim % i_channels;
		int s = i % spatial_dim;
		
		b[(n*channels+cur_channels+c)*spatial_dim+s] = a[i];
	}
}

template <typename Dtype>
void DeConcatLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
	int num = bottom[0]->num();
	int channels = bottom[0]->channels();
	int height = bottom[0]->height();
	int width = bottom[0]->width();
	
	int cur_channels = 0;
	for (int i =0; i < top.size();i++)
	{
		int i_channels = top[i]->channels();
		deconcat_forward<Dtype><<<CAFFE_GET_BLOCKS(top[i]->count()), CAFFE_CUDA_NUM_THREADS>>>
		(top[i]->count(),channels,i_channels,cur_channels,height*width,
					bottom[0]->gpu_data(), top[i]->mutable_gpu_data());		
		cur_channels += i_channels;
	}
}

template <typename Dtype>
void DeConcatLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, const vector<Blob<Dtype>*>& bottom) 
{
	int num = bottom[0]->num();
	int channels = bottom[0]->channels();
	int height = bottom[0]->height();
	int width = bottom[0]->width();
	
	int cur_channels = 0;
	for (int i =0; i < top.size();i++)
	{
		int i_channels = top[i]->channels();
		deconcat_backward<Dtype><<<CAFFE_GET_BLOCKS(top[i]->count()), CAFFE_CUDA_NUM_THREADS>>>
		(top[i]->count(),channels,i_channels,cur_channels,height*width,
					top[i]->gpu_diff(), bottom[0]->mutable_gpu_diff());		
		cur_channels += i_channels;
	}
}
template <typename Dtype>
void DeConcatLayer<Dtype>::SecForward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
}
INSTANTIATE_LAYER_GPU_FUNCS(DeConcatLayer);
}  // namespace caffe
