#include "hip/hip_runtime.h"

#include <vector>

#include "caffe/layers/loss/gan_softmax_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/solver.hpp"
namespace caffe {

template <typename Dtype>
static __global__ void forward_kernel(const int num_spatial, const int num, const int channels, const int spatial_dim,
          const Dtype* prob_data, const Dtype* label,  Dtype* loss) 
{
  CUDA_KERNEL_LOOP(index, num_spatial) 
  {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    int label_value = static_cast<int>(label[index]);
   		

    loss[index] = -log(prob_data[(n * channels + label_value) * spatial_dim + s]);
  }
}

template <typename Dtype>
static __global__ void backward_kernel(const int num_spatial,const int num, const int channels, const int spatial_dim, 
					const Dtype* prob_data, const Dtype* label,  Dtype* bottom_diff) 
{
  CUDA_KERNEL_LOOP(index, num_spatial) 
  {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    int label_value = static_cast<int>(label[index]);
		
  	for (int c = 0; c < channels; ++c) 
    {
    	int ind = (n*channels+c)*spatial_dim+s;
    	if (c == label_value)
    		bottom_diff[ind] = prob_data[ind] -1;
			else
				bottom_diff[ind] = prob_data[ind];
    }
  }
}

template <typename Dtype>
void GANSoftmaxWithLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
	softmax_layer_->Forward_gpu(softmax_bottom_vec_, softmax_top_vec_);

 
 	if (Caffe::gan_type() == "train_dnet")
 	{	
		int num = bottom[0]->num()/2;
		int channels = bottom[0]->channels();
		int height = bottom[0]->height();
		int width = bottom[0]->width();

		for (int i=0;i<bottom[1]->count();i++)
		{
			CHECK_GE(bottom[1]->cpu_data()[i],0);
			CHECK_LE(bottom[1]->cpu_data()[i],channels-1);
		}
 	
 		
		forward_kernel<Dtype><<<CAFFE_GET_BLOCKS(num*height*width), CAFFE_CUDA_NUM_THREADS>>>
		(num*height*width, num, channels, height*width, prob_.gpu_data()+prob_.offset(num), bottom[1]->gpu_data(), loss_.mutable_gpu_data());
		
		Dtype loss;
		caffe_gpu_asum(num*height*width, loss_.gpu_data(), &loss);

		top[0]->mutable_cpu_data()[0] = loss / loss_.count() * Dtype(1);
	}
	else
	{
		int num = bottom[0]->num();
		int channels = bottom[0]->channels();
		int height = bottom[0]->height();
		int width = bottom[0]->width();

		for (int i=0;i<bottom[1]->count();i++)
		{
			CHECK_GE(bottom[1]->cpu_data()[i],0);
			CHECK_LE(bottom[1]->cpu_data()[i],channels-1);
		}
 		
 	
		forward_kernel<Dtype><<<CAFFE_GET_BLOCKS(num*height*width), CAFFE_CUDA_NUM_THREADS>>>
		(num*height*width, num, channels, height*width, prob_.gpu_data(), bottom[1]->gpu_data(), loss_.mutable_gpu_data());
		
		Dtype loss;
		caffe_gpu_asum(num*height*width, loss_.gpu_data(), &loss);

		top[0]->mutable_cpu_data()[0] = loss / loss_.count() * Dtype(0.1);
	}
	
	if (Solver<Dtype>::iter() % 100 == 0 && Caffe::gan_type() == "train_dnet")
			LOG(INFO)<<"softmax_loss = "<<top[0]->cpu_data()[0];
}

template <typename Dtype>
void GANSoftmaxWithLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, const vector<Blob<Dtype>*>& bottom) 
{
	if (Caffe::gan_type() == "train_dnet")
 	{	
 		int num = bottom[0]->num()/2;
		int channels = bottom[0]->channels();
		int height = bottom[0]->height();
		int width = bottom[0]->width();
		
		
		caffe_gpu_set(bottom[0]->count(),Dtype(0),bottom[0]->mutable_gpu_diff());
		backward_kernel<Dtype><<<CAFFE_GET_BLOCKS(num*height*width), CAFFE_CUDA_NUM_THREADS>>>
		(num*height*width, num, channels, height*width, prob_.gpu_data()+prob_.offset(num), bottom[1]->gpu_data(), 
																					bottom[0]->mutable_gpu_diff()+bottom[0]->offset(num));


	
		const Dtype loss_weight =  top[0]->cpu_diff()[0] / loss_.count() * Dtype(1);
		caffe_gpu_scal(prob_.count(), loss_weight, bottom[0]->mutable_gpu_diff());
 	}
 	else
 	{
		int num = bottom[0]->num();
		int channels = bottom[0]->channels();
		int height = bottom[0]->height();
		int width = bottom[0]->width();
	
	

		backward_kernel<Dtype><<<CAFFE_GET_BLOCKS(num*height*width), CAFFE_CUDA_NUM_THREADS>>>
		(num*height*width, num, channels, height*width, prob_.gpu_data(), bottom[1]->gpu_data(), bottom[0]->mutable_gpu_diff());


		const Dtype loss_weight = top[0]->cpu_diff()[0] / loss_.count() * Dtype(0.1);
		caffe_gpu_scal(prob_.count(), loss_weight, bottom[0]->mutable_gpu_diff());
	}
}

template <typename Dtype>
void GANSoftmaxWithLossLayer<Dtype>::SecForward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
}

INSTANTIATE_LAYER_GPU_FUNCS(GANSoftmaxWithLossLayer);
}  // namespace caffe
