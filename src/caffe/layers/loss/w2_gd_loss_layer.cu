#include "hip/hip_runtime.h"
#include <vector>
#include "caffe/solver.hpp"
#include "caffe/layers/loss/w2_gd_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include <cfloat>
namespace caffe {
//-----------------------------------
template <typename Dtype>
static __global__ void compute_sum(int num_spatial,int num, int channels, int spatial_dim, const Dtype* bottom_sec_diff, const Dtype * prob, Dtype* sum) 
{
  CUDA_KERNEL_LOOP(i, num_spatial) 
  {
  	int n = i / spatial_dim;
  	int s = i % spatial_dim;
  	
  	Dtype temp = 0;
  	for (int iter = 0; iter<channels-1; iter++)
  	{
  		int index = (n*channels+iter) * spatial_dim + s;
  		temp += bottom_sec_diff[index] * prob[index];
  	}
  	sum[i] = temp;
  }
}
template <typename Dtype>
static __global__ void secforward_kernel(const int count, const int num, const int channels, const int spatial_dim, 
					const Dtype* prob, const Dtype* label, const Dtype* bottom_sec_diff, const Dtype* sum_secx_p,  Dtype* bottom_diff) 
{
  CUDA_KERNEL_LOOP(index, count) 
  {
  	const int n = index / spatial_dim / channels;
  	const int c = index / spatial_dim % channels;
    const int s = index % spatial_dim;
    
    if (c <  channels-1)
  		bottom_diff[index] = bottom_sec_diff[index]*prob[index] - sum_secx_p[n*spatial_dim+s] *  prob[index];
  	else
  		bottom_diff[index] = 0;
  }
}
//-----------------------------------
template <typename Dtype>
static __global__ void Dloss_forward_kernel(int count, int num,int channels, int spatial_dim, const Dtype *in, const Dtype *label, 
																				Dtype * prob, Dtype *loss_g, Dtype * loss_d, Dtype *loss_c)
{
	CUDA_KERNEL_LOOP(i, count)
	{	
		int n = i / spatial_dim;
		int s = i % spatial_dim;
		
		loss_g[n*spatial_dim+s] = -in[(n*channels+channels-1)*spatial_dim+s]; 
		loss_d[n*spatial_dim+s] = -in[((n+num)*channels+channels-1)*spatial_dim+s];
		
		#if 1
		Dtype max_value = in[(n*channels+0)*spatial_dim+s];
		for (int iter=0;iter<channels-1;iter++)
			max_value = max(max_value,in[(n*channels+iter)*spatial_dim+s]);			
		Dtype sum = 0;
		int label_index = label[n*spatial_dim+s];		
		for (int iter=0;iter<channels-1;iter++)
			sum += exp(in[(n*channels+iter)*spatial_dim+s]-max_value);
		for (int iter=0;iter<channels-1;iter++)
			prob[(n*channels+iter)*spatial_dim+s] = exp(in[(n*channels+iter)*spatial_dim+s]-max_value) / sum;
		loss_c[n*spatial_dim+s] = -log(max(prob[(n*channels+label_index)*spatial_dim+s],Dtype(FLT_MIN)));
		
		max_value = in[((n+num)*channels+0)*spatial_dim+s];
		for (int iter=0;iter<channels-1;iter++)
			max_value = max(max_value,in[((n+num)*channels+iter)*spatial_dim+s]);
			
		sum = 0;
		for (int iter=0;iter<channels-1;iter++)
			sum += exp(in[((n+num)*channels+iter)*spatial_dim+s]-max_value);
		for (int iter=0;iter<channels-1;iter++)
			prob[((n+num)*channels+iter)*spatial_dim+s] = exp(in[((n+num)*channels+iter)*spatial_dim+s]-max_value) / sum;
		loss_c[(n+num)*spatial_dim+s] = -log(max(prob[((n+num)*channels+label_index)*spatial_dim+s],Dtype(FLT_MIN)));
		#endif
	}
}
template <typename Dtype>
static __global__ void Gloss_forward_kernel(int count, int num,int channels, int spatial_dim, const Dtype *in, const Dtype *label, 
																				Dtype * prob, Dtype *loss_g, Dtype * loss_c)
{
	CUDA_KERNEL_LOOP(i, count)
	{	
		int n = i / spatial_dim;
		int s = i % spatial_dim;
		
		loss_g[n*spatial_dim+s] = -in[(n*channels+channels-1)*spatial_dim+s]; 
		
		#if 1
		Dtype max_value = in[(n*channels+0)*spatial_dim+s];
		for (int iter=0;iter<channels-1;iter++)
			max_value = max(max_value,in[(n*channels+iter)*spatial_dim+s]);			
		Dtype sum = 0;
		int label_index = label[n*spatial_dim+s];		
		for (int iter=0;iter<channels-1;iter++)
			sum += exp(in[(n*channels+iter)*spatial_dim+s]-max_value);
		for (int iter=0;iter<channels-1;iter++)
			prob[(n*channels+iter)*spatial_dim+s] = exp(in[(n*channels+iter)*spatial_dim+s]-max_value) / sum;
		loss_c[n*spatial_dim+s] = -log(max(prob[(n*channels+label_index)*spatial_dim+s],Dtype(FLT_MIN)));
		#endif
	}
}

template <typename Dtype>
static __global__ void Dloss_backward_kernel(int count, int num,int channels, int spatial_dim, const Dtype *data_in,const Dtype *label, const Dtype *prob,
																					Dtype *diff_in)
{
	CUDA_KERNEL_LOOP(i, count)
	{	
		int n = i / spatial_dim;
		int s = i % spatial_dim;
		
		diff_in[(n*channels+channels-1)*spatial_dim+s] = 1;
		diff_in[((n+num)*channels+channels-1)*spatial_dim+s] = -1;
#if 1
		for (int iter=0;iter<channels-1;iter++)
		{
			diff_in[(n*channels+iter)*spatial_dim+s] = 1 * prob[(n*channels+iter)*spatial_dim+s];
			diff_in[((n+num)*channels+iter)*spatial_dim+s] = 1 * prob[((n+num)*channels+iter)*spatial_dim+s];
		}	
		
		int label_index = label[n*spatial_dim+s];
		diff_in[(n*channels+label_index)*spatial_dim+s] -= 1;
		
		label_index = label[n*spatial_dim+s];
		diff_in[((n+num)*channels+label_index)*spatial_dim+s] -= 1;
#endif
	}
}
template <typename Dtype>
static __global__ void Gloss_backward_kernel(int count, int num,int channels, int spatial_dim, const Dtype *data_in,const Dtype *label, const Dtype *prob,
																					Dtype *diff_in)
{
	CUDA_KERNEL_LOOP(i, count)
	{		
		int n = i / spatial_dim;
		int s = i % spatial_dim;
		
		diff_in[(n*channels+channels-1)*spatial_dim+s] = -1;
#if 1
		for (int iter=0;iter<channels-1;iter++)
		{
			diff_in[(n*channels+iter)*spatial_dim+s] = 0.5 * 0.1 * prob[(n*channels+iter)*spatial_dim+s];
		}	
		
		int label_index = label[n*spatial_dim+s];
		diff_in[(n*channels+label_index)*spatial_dim+s] -= 0.5 * 0.1;
#endif
	}
}

template <typename Dtype>
void W2GdLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
	Dtype loss_g, loss_d, loss_c;
	if (Caffe::gan_type() == "train_dnet")
	{	
		int num = bottom[0]->num();
		int channels = bottom[0]->channels();
		int height = bottom[0]->height();
		int width = bottom[0]->width();
	
	
		Dloss_forward_kernel<Dtype><<<CAFFE_GET_BLOCKS(num/2*height*width), CAFFE_CUDA_NUM_THREADS>>>
		(num/2*height*width, num/2, channels, height*width, bottom[0]->gpu_data(), bottom[1]->gpu_data(), 
		prob_.mutable_gpu_data(), loss_g_.mutable_gpu_data(),loss_d_.mutable_gpu_data(), loss_c_.mutable_gpu_data());	

		caffe_gpu_sum(loss_g_.count(),loss_g_.gpu_data(),top[0]->mutable_gpu_data());	
		loss_g = top[0]->cpu_data()[0];	
			
		caffe_gpu_sum(loss_d_.count(),loss_d_.gpu_data(),top[0]->mutable_gpu_data());	
		loss_d = top[0]->cpu_data()[0];	
		
		caffe_gpu_sum(loss_c_.count(),loss_c_.gpu_data(),top[0]->mutable_gpu_data());	
		loss_c = top[0]->cpu_data()[0];	
		top[0]->mutable_cpu_data()[0] = loss_d / Dtype(num/2*height*width) - loss_g / Dtype(num/2*height*width) + loss_c / Dtype(num*height*width);
	}
	else
	{
		int num = bottom[0]->num();
		int channels = bottom[0]->channels();
		int height = bottom[0]->height();
		int width = bottom[0]->width();
		
		Gloss_forward_kernel<Dtype><<<CAFFE_GET_BLOCKS(num*height*width), CAFFE_CUDA_NUM_THREADS>>>
		(num*height*width, num, channels, height*width, bottom[0]->gpu_data(), bottom[1]->gpu_data(), 
		prob_.mutable_gpu_data(), loss_g_.mutable_gpu_data(), loss_c_.mutable_gpu_data());	

		caffe_gpu_sum(loss_g_.count(),loss_g_.gpu_data(),top[0]->mutable_gpu_data());			
		loss_g = 	top[0]->cpu_data()[0];		
		
		caffe_gpu_sum(loss_c_.count(),loss_c_.gpu_data(),top[0]->mutable_gpu_data());	
		loss_c = top[0]->cpu_data()[0];	
		top[0]->mutable_cpu_data()[0] = loss_g / Dtype(num*height*width) + 0.1 * loss_c / Dtype(num*height*width);
	}
}

template <typename Dtype>
void W2GdLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, const vector<Blob<Dtype>*>& bottom) 
{
	if (Caffe::second_pass() == false)
	{
		if (Caffe::gan_type() == "train_dnet")
		{
			int num = bottom[0]->num();
			int channels = bottom[0]->channels();
			int height = bottom[0]->height();
			int width = bottom[0]->width();
		
			Dtype loss_weights_ = top[0]->cpu_diff()[0] / Dtype(num/2*1*height*width);
			
			Dloss_backward_kernel<Dtype><<<CAFFE_GET_BLOCKS(num/2*height*width), CAFFE_CUDA_NUM_THREADS>>>
			(num/2*height*width, num/2,channels,height*width, bottom[0]->gpu_data(), bottom[1]->gpu_data(), prob_.gpu_data(),
			bottom[0]->mutable_gpu_diff());	
			
			caffe_gpu_scal(bottom[0]->count(),loss_weights_,bottom[0]->mutable_gpu_diff());		
		}
		else
		{
			int num = bottom[0]->num();
			int channels = bottom[0]->channels();
			int height = bottom[0]->height();
			int width = bottom[0]->width();
		
			Dtype loss_weights_ = top[0]->cpu_diff()[0] / Dtype(num*1*height*width);
			
			Gloss_backward_kernel<Dtype><<<CAFFE_GET_BLOCKS(num*height*width), CAFFE_CUDA_NUM_THREADS>>>
			(num*height*width, num,channels,height*width, bottom[0]->gpu_data(),bottom[1]->gpu_data(), prob_.gpu_data(),
			bottom[0]->mutable_gpu_diff());	
			
			caffe_gpu_scal(bottom[0]->count(),loss_weights_,bottom[0]->mutable_gpu_diff());	
		}
	}
	else
	{
	}
}
template <typename Dtype>
void W2GdLossLayer<Dtype>::SecForward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
#if 0
	int num = bottom[0]->num();
  int channels = bottom[0]->channels();
  int height = bottom[0]->height();
  int width = bottom[0]->width();
  
  compute_sum<Dtype><<<CAFFE_GET_BLOCKS(num*height*width), CAFFE_CUDA_NUM_THREADS>>>
  (num*height*width, num, channels, height*width, bottom[0]->gpu_sec_diff(), prob_.gpu_data(), loss_c_.mutable_gpu_data()); 

	secforward_kernel<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>
	(bottom[0]->count(), num, channels, height*width, prob_.gpu_data(), bottom[1]->gpu_data(), bottom[0]->gpu_sec_diff(), loss_c_.gpu_data(),
	bottom[0]->mutable_gpu_diff());

	const Dtype loss_weight = top[0]->cpu_diff()[0] / Dtype(num/2*channels*height*width) * 1;
	caffe_gpu_scal(bottom[0]->count(), loss_weight, bottom[0]->mutable_gpu_diff()); 
#endif	
}
INSTANTIATE_LAYER_GPU_FUNCS(W2GdLossLayer);
}  // namespace caffe
