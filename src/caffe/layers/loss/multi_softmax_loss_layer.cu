#include "hip/hip_runtime.h"

#include <vector>

#include "caffe/layers/loss/multi_softmax_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
static __global__ void forward_kernel(const int num_spatial, const int num, const int channels, const int spatial_dim,
          const Dtype* prob_data, const Dtype* label,  Dtype* loss) 
{
  CUDA_KERNEL_LOOP(index, num_spatial) 
  {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    int label_value = static_cast<int>(label[index]);
   		

    loss[index] = -log(prob_data[(n * channels + label_value) * spatial_dim + s]);
  }
}

template <typename Dtype>
static __global__ void backward_kernel(const int num_spatial,const int num, const int channels, const int spatial_dim, 
					const Dtype* prob_data, const Dtype* label,  Dtype* bottom_diff) 
{
  CUDA_KERNEL_LOOP(index, num_spatial) 
  {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    int label_value = static_cast<int>(label[index]);
		
  	for (int c = 0; c < channels; ++c) 
    {
    	int ind = (n*channels+c)*spatial_dim+s;
    	if (c == label_value)
    		bottom_diff[ind] = prob_data[ind] - 1;
			else
				bottom_diff[ind] = prob_data[ind];
    }
  }
}


template <typename Dtype>
void MultiSoftmaxWithLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
	softmax_layer_->Forward_gpu(softmax_bottom_vec_, softmax_top_vec_);
	
	int num = bottom[0]->num();
	int channels = bottom[0]->channels();
	int height = bottom[0]->height();
	int width = bottom[0]->width();
	
	//LOG(INFO)<<"num = "<<num<<", channels = "<<channels;
	
	forward_kernel<Dtype><<<CAFFE_GET_BLOCKS(num*height*width), CAFFE_CUDA_NUM_THREADS>>>
	(num*height*width, num, channels, height*width, prob_.gpu_data(), bottom[1]->gpu_data(), loss_.mutable_gpu_data());
	
	Dtype loss;
	caffe_gpu_asum(num*height*width, loss_.gpu_data(), &loss);

	top[0]->mutable_cpu_data()[0] = loss / loss_.count();
}

template <typename Dtype>
void MultiSoftmaxWithLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, const vector<Blob<Dtype>*>& bottom) 
{
	int num = bottom[0]->num();
	int channels = bottom[0]->channels();
	int height = bottom[0]->height();
	int width = bottom[0]->width();
	
	caffe_gpu_set(bottom[0]->count(),Dtype(0),bottom[0]->mutable_gpu_diff());
	backward_kernel<Dtype><<<CAFFE_GET_BLOCKS(num*height*width), CAFFE_CUDA_NUM_THREADS>>>
	(num*height*width, num, channels, height*width, prob_.gpu_data(), bottom[1]->gpu_data(), 
																				bottom[0]->mutable_gpu_diff());

	const Dtype loss_weight =  top[0]->cpu_diff()[0] / loss_.count();
	caffe_gpu_scal(prob_.count(), loss_weight, bottom[0]->mutable_gpu_diff());
}

template <typename Dtype>
void MultiSoftmaxWithLossLayer<Dtype>::SecForward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
}

INSTANTIATE_LAYER_GPU_FUNCS(MultiSoftmaxWithLossLayer);
}  // namespace caffe
