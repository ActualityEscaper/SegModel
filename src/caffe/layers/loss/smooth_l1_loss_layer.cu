#include "hip/hip_runtime.h"
#include "caffe/layers/loss/smooth_l1_loss_layer.hpp"

namespace caffe {

template <typename Dtype>
static __global__ void SmoothL1Forward(const int n, const int ignore_value, const Dtype* in_0, const Dtype * in_1, Dtype* out, Dtype * count) 
{
  CUDA_KERNEL_LOOP(index, n) 
  {
  	if (in_0[index] == ignore_value)
  	{
  		count[index] = 0;
  		out[index] = 0;
  	}
  	else
  	{
			count[index] = 1;
			
		  Dtype val = abs(in_0[index] - in_1[index]);
			
		  out[index] = val;
		}    
  }
}
template <typename Dtype>
static __global__ void SmoothL1Backward(const int n, const int ignore_value, const Dtype* in_0, const Dtype * in_1, Dtype* in_0_diff, Dtype * count) 
{
  CUDA_KERNEL_LOOP(index, n) 
  {
  	if (in_0[index] == ignore_value)
  	{
  		count[index] = 0;
  		in_0_diff[index] = 0;
  	}
  	else
  	{
		  Dtype val = in_0[index] - in_1[index];
		  if (val > 0)
		  	in_0_diff[index] = Dtype(1);
		  else
		  	in_0_diff[index] = Dtype(-1);
		}    
  }
}

template <typename Dtype>
void SmoothL1LossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
  
  SmoothL1Forward<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>
  (bottom[0]->count(),ignore_value,bottom[0]->gpu_data(),bottom[1]->gpu_data(), loss_.mutable_gpu_data(),counts_.mutable_gpu_data());

	Dtype counts;
  caffe_gpu_asum(counts_.count(), counts_.gpu_data(), &counts);
  
  Dtype loss;
  caffe_gpu_asum(loss_.count(), loss_.gpu_data(), &loss);
  
  top[0]->mutable_cpu_data()[0] = loss / counts;
}

template <typename Dtype>
void SmoothL1LossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, const vector<Blob<Dtype>*>& bottom) 
{
  SmoothL1Backward<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>
  (bottom[0]->count(),ignore_value,bottom[0]->gpu_data(),bottom[1]->gpu_data(), bottom[0]->mutable_gpu_diff(),counts_.mutable_gpu_data());
  
  Dtype counts;
  caffe_gpu_asum(counts_.count(), counts_.gpu_data(), &counts);
  
  caffe_gpu_scal(bottom[0]->count(),top[0]->cpu_diff()[0] / counts,bottom[0]->mutable_gpu_diff());
}
template <typename Dtype>
void SmoothL1LossLayer<Dtype>::SecForward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
}
INSTANTIATE_LAYER_GPU_FUNCS(SmoothL1LossLayer);

}  // namespace caffe
