#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/loss/gd_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
static __global__ void Dloss_forward_kernel(int count, const Dtype *in, Dtype *out)
{

	CUDA_KERNEL_LOOP(i, count)
	{		
		out[i] = max(in[i],Dtype(0))   + log(Dtype(1)+exp(-abs(in[i])))
					 + max(in[i+count],Dtype(0)) - in[i+count] + log(Dtype(1)+exp(-abs(in[i+count])));			
	}
}

template <typename Dtype>
static __global__ void Gloss_forward_kernel(int count, const Dtype *in, Dtype *out)
{
	CUDA_KERNEL_LOOP(i, count)
	{	
		out[i] =  max(in[i],Dtype(0))-in[i]+log(Dtype(1)+exp(-abs(in[i])));	
	}
}
template <typename Dtype>
static __global__ void Dloss_backward_kernel(int count, const Dtype *data_in, Dtype *diff_in)
{

	CUDA_KERNEL_LOOP(i, count)
	{	
		if (data_in[i] > 0) 
			diff_in[i] =  Dtype(1.0) / (Dtype(1)+exp(-abs(data_in[i])));
		else	
			diff_in[i] = 1 -  Dtype(1.0) / (Dtype(1)+exp(-abs(data_in[i])));
			
			
		if (data_in[i+count] > 0) 
			diff_in[i+count] = - 1 + Dtype(1.0) / (Dtype(1)+exp(-abs(data_in[i+count])));
		else
			diff_in[i+count] = - Dtype(1.0) / (Dtype(1)+exp(-abs(data_in[i+count])));
	}
}
template <typename Dtype>
static __global__ void Gloss_backward_kernel(int count, const Dtype *data_in, Dtype *diff_in)
{

	CUDA_KERNEL_LOOP(i, count)
	{		
		if (data_in[i] > 0) 
			diff_in[i] = -1 + Dtype(1.0) / (Dtype(1)+exp(-abs(data_in[i])));
		else
			diff_in[i] =  - Dtype(1.0) / (Dtype(1)+exp(-abs(data_in[i])));	
		diff_in[i+count] = 0;			
	}
}
//---------------------------------
template <typename Dtype>
static __global__ void Dloss_secforward_kernel(int count, const Dtype *in_sec_diff, const Dtype * in_data, Dtype *in_diff)
{
	CUDA_KERNEL_LOOP(i, count)
	{	
		in_diff[i] =  in_sec_diff[i] * 	exp(-in_data[i]) / ( (Dtype(1)+exp(-in_data[i]))*(Dtype(1)+exp(-in_data[i])) );
		in_diff[i+count] = in_sec_diff[i+count] * exp(in_data[i+count]) / ( (Dtype(1)+exp(in_data[i+count]))*(Dtype(1)+exp(in_data[i+count])) );
	}
}
//---------------------------------
template <typename Dtype>
void GdLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
	int num = bottom[0]->num();
  int channels = bottom[0]->channels();
  int height = bottom[0]->height();
  int width = bottom[0]->width();

	
	CHECK_EQ(bottom.size(),1);
	CHECK_EQ(num%2,0);
	CHECK_EQ(channels,1);

	if (Caffe::gan_type() == "train_dnet")
	{	
		Dloss_forward_kernel<Dtype><<<CAFFE_GET_BLOCKS(num/2*height*width), CAFFE_CUDA_NUM_THREADS>>>
		(num/2*height*width,bottom[0]->gpu_data(),loss_.mutable_gpu_data());	
	}
	else
	{
		Gloss_forward_kernel<Dtype><<<CAFFE_GET_BLOCKS(num/2*height*width), CAFFE_CUDA_NUM_THREADS>>>
		(num/2*height*width,bottom[0]->gpu_data(),loss_.mutable_gpu_data());	
	}
	Dtype sum;
	caffe_gpu_asum(loss_.count(),loss_.gpu_data(),&sum);
	top[0]->mutable_cpu_data()[0] = sum / (num/2*height*width);
}

template <typename Dtype>
void GdLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, const vector<Blob<Dtype>*>& bottom) 
{
	int num = bottom[0]->num();
  int channels = bottom[0]->channels();
  int height = bottom[0]->height();
  int width = bottom[0]->width();
	if (Caffe::second_pass() == false)
	{	
		if (Caffe::gan_type() == "train_dnet")
		{
			Dloss_backward_kernel<Dtype><<<CAFFE_GET_BLOCKS(num/2*height*width), CAFFE_CUDA_NUM_THREADS>>>
			(num/2*height*width,bottom[0]->gpu_data(),bottom[0]->mutable_gpu_diff());			
		}
		else
		{
			Gloss_backward_kernel<Dtype><<<CAFFE_GET_BLOCKS(num/2*height*width), CAFFE_CUDA_NUM_THREADS>>>
			(num/2*height*width,bottom[0]->gpu_data(),bottom[0]->mutable_gpu_diff());			
		}		
		Dtype loss_weights_ = top[0]->cpu_diff()[0] / (num/2*height*width);
		caffe_gpu_scal(bottom[0]->count(),loss_weights_,bottom[0]->mutable_gpu_diff());		
	}
	else
	{	
	}
}
template <typename Dtype>
void GdLossLayer<Dtype>::SecForward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
	int num = bottom[0]->num();
  int channels = bottom[0]->channels();
  int height = bottom[0]->height();
  int width = bottom[0]->width();
  
	Dloss_secforward_kernel<Dtype><<<CAFFE_GET_BLOCKS(num/2*height*width), CAFFE_CUDA_NUM_THREADS>>>
	(num/2*height*width,bottom[0]->gpu_sec_diff(),bottom[0]->gpu_data(),bottom[0]->mutable_gpu_diff());	
	
	Dtype loss_weights_ = top[0]->cpu_diff()[0] / (num/2*height*width);
	caffe_gpu_scal(bottom[0]->count(),loss_weights_,bottom[0]->mutable_gpu_diff());	
}
INSTANTIATE_LAYER_GPU_FUNCS(GdLossLayer);
}  // namespace caffe
