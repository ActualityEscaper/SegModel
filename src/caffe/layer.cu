#include "hip/hip_runtime.h"
#include "caffe/layer.hpp"
#include "caffe/solver.hpp"
#include<cfloat>
namespace caffe {
template <typename Dtype>
static __global__ void scale_kernel(int count, int image_dim, Dtype sec_loss_weight, Dtype norm_value, 
																			const Dtype *in, const Dtype *coef, Dtype *out)
{

	CUDA_KERNEL_LOOP(i, count)
	{
		int n = i / image_dim;
		out[i] = 2 * sec_loss_weight  *(coef[n]-norm_value)/ coef[n] * in[i];	
	} 
}
template <typename Dtype>
static __global__ void compute_sum(int image_dim, const Dtype *in, Dtype *out)
{
	__shared__ Dtype buffer[CAFFE_CUDA_NUM_THREADS];

	buffer[threadIdx.x] = 0;
	for (int i = threadIdx.x;i < image_dim;i += blockDim.x)
		buffer[threadIdx.x] += in[blockIdx.x*image_dim+i]*in[blockIdx.x*image_dim+i];
	__syncthreads();
	
	for (int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (threadIdx.x < s)
			buffer[threadIdx.x] += buffer[threadIdx.x+s];
		__syncthreads();
	}
	
	if (threadIdx.x == 0)
		out[blockIdx.x] = sqrt(buffer[0]);
}
template <typename Dtype>
void Layer<Dtype>::compute_sec_loss(const vector<Blob<Dtype>*>& top, const Dtype sec_loss_weight, const Dtype norm_value)
{
	vector<shared_ptr<Blob<Dtype> > > sum_;
	sum_.resize(top.size());
	for (int i=0;i < top.size();i++)
	{
		CUDA_CHECK(hipSetDevice(Caffe::GPUs[i%NGPUS]));
		int num = top[i]->num();
		int channels = top[i]->channels();
		int height = top[i]->height();
		int width = top[i]->width();	
		
		sum_[i].reset(new Blob<Dtype>(num,1,1,1));
		compute_sum<<<num,CAFFE_CUDA_NUM_THREADS>>>
		(channels*height*width,top[i]->gpu_diff(),sum_[i]->mutable_gpu_data());
		
		if (Solver<Dtype>::iter() % 1000 == 0)
		{
			Dtype sum = 0;
			for (int iter = 0;iter<num;iter++)
				sum += sum_[i]->cpu_data()[iter];
			LOG(INFO)<<"sum = "<<sum/Dtype(num);
		}
		scale_kernel<Dtype><<<CAFFE_GET_BLOCKS(top[i]->count()), CAFFE_CUDA_NUM_THREADS>>>
		(top[i]->count(), channels*height*width, sec_loss_weight, norm_value,
		top[i]->gpu_diff(), sum_[i]->gpu_data(), top[i]->mutable_gpu_sec_diff());	
		
		caffe_gpu_scal(top[i]->count(),Dtype(1)/Dtype(num),top[i]->mutable_gpu_sec_diff());
	}
	CUDA_CHECK(hipSetDevice(Caffe::GPUs[0]));
}
//----------------------------------------- proto <->  memory--------------------
template <typename Dtype>
void Layer<Dtype>::ToProto(LayerParameter* param, bool write_diff) 
{
  param->Clear();
  param->CopyFrom(layer_param_);
  param->clear_blobs();
  for (int i = 0; i < blobs_.size(); ++i) 
    blobs_[i]->ToProto(param->add_blobs(), write_diff);
  
}

template <typename Dtype>
inline Dtype Layer<Dtype>::Forward(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top)
{
	//LOG(INFO)<<"-----------processing "<<this->layer_param_.type()<<", top.size() = "<<top.size();

	Forward_gpu(bottom, top);

	Dtype loss_weight = layer_param_.include().loss_weight();
	
	Dtype loss = 0;
	if (loss_weight > 0)
	{
		CHECK_EQ(Caffe::GPUs.size(),top.size());
		for (int i=0;i<top.size();i++)
			loss += top[i]->cpu_data()[0] * loss_weight / Dtype(top.size());
	}

	return loss;
};


template <typename Dtype>
inline void Layer<Dtype>::Backward(const vector<Blob<Dtype>*>& top, const vector<Blob<Dtype>*>& bottom) 
{
	Dtype loss_weight = layer_param_.include().loss_weight();
	if (loss_weight > 0)
	{
		CHECK_EQ(Caffe::GPUs.size(),top.size());	
		for (int i=0;i<top.size();i++)
			top[i]->mutable_cpu_diff()[0] = loss_weight / Dtype(top.size());
	}
	
  Backward_gpu(top, bottom);	
};

template <typename Dtype>
inline void Layer<Dtype>::SecForward(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top)
{
	//LOG(INFO)<<"-----------processing "<<this->layer_param_.type()<<", top.size() = "<<top.size();
	Dtype sec_loss_weight = layer_param_.include().sec_loss_weight();
	Dtype norm_value = layer_param_.include().norm_value();
	
	if (sec_loss_weight > 0 && Caffe::second_pass())
	{
		CHECK_EQ(Caffe::GPUs.size(),top.size());	
		compute_sec_loss(top,sec_loss_weight / Dtype(top.size()), norm_value);
	}
	else
	{
		SecForward_gpu(bottom, top);
	}
};

INSTANTIATE_CLASS(Layer);
}
